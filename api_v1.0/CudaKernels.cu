#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

__global__ void grayscaleKernel(unsigned char* input, unsigned char* output, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width && y < height) {
        int idx = (y * width + x) * 3;
        unsigned char r = input[idx];
        unsigned char g = input[idx + 1];
        unsigned char b = input[idx + 2];
        unsigned char gray = (unsigned char)(0.299f * r + 0.587f * g + 0.114f * b);
        output[y * width + x] = gray;
    }
}

__global__ void upsampleKernel(unsigned char* input, unsigned char* output, int inputWidth, int inputHeight, int scaleFactor) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < inputWidth * scaleFactor && y < inputHeight * scaleFactor) {
        int srcX = x / scaleFactor;
        int srcY = y / scaleFactor;

        for (int c = 0; c < 3; c++) {
            output[(y * inputWidth * scaleFactor + x) * 3 + c] = input[(srcY * inputWidth + srcX) * 3 + c];
        }
    }
}

extern "C" void launchGrayscaleKernel(unsigned char* input, unsigned char* output, int width, int height, hipStream_t stream) {
    dim3 blockSize(16, 16);
    dim3 gridSize((width + 15) / 16, (height + 15) / 16);
    grayscaleKernel<<<gridSize, blockSize, 0, stream>>>(input, output, width, height);
    hipDeviceSynchronize();
    std::cout << "Grayscale kernel execution complete." << std::endl;
}

extern "C" void performFFTKernel(float* input, hipfftComplex* output, int width, int height, hipStream_t stream) {
    hipfftHandle plan;
    hipfftResult result;
    std::cout << "FFT KERNEL START..." << std::endl;
    result = hipfftPlan2d(&plan, width, height, HIPFFT_R2C);
    result = hipfftSetStream(plan, stream);
    result = hipfftExecR2C(plan, input, output);
    result = hipfftDestroy(plan);
    hipError_t cudaResult = hipDeviceSynchronize();
    std::cout << "FFT KERNEL END..." << std::endl;
}

extern "C" void launchUpsampleKernel(unsigned char* input, unsigned char* output, int inputWidth, int inputHeight, int scaleFactor, hipStream_t stream) {
    // purely for warp of 32 alignment? -im not sure if this is right, the output is losing quality. 
    dim3 blockSize(16, 16);
    dim3 gridSize((inputWidth * scaleFactor + 15) / 16, (inputHeight * scaleFactor + 15) / 16);

    upsampleKernel<<<gridSize, blockSize, 0, stream>>>(input, output, inputWidth, inputHeight, scaleFactor);
    // do we need this?
    hipDeviceSynchronize();
    std::cout << "Upsampling kernel execution complete." << std::endl;
}
